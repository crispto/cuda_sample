#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#define N 1024 * 1024

void check(int ret, const char *const func_name, const char *file, const int line_num)
{
    if (ret)
    {
        fprintf(stderr, "error [%s:%d] func_name: %s, code: %d\n", file, line_num, func_name, ret);
    }
}

#define my_check_error(val) check((val), #val, __FILE__, __LINE__)
int init_cuda(int &clock_rate)
{
    int count;
    hipGetDeviceCount(&count);
    printf("device count: %d\n", count);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("name: %s, major %d, mino: %d, total mem: %ld\n", prop.name, prop.major, prop.minor, prop.totalGlobalMem / (1024 * 1024));
    clock_rate = prop.clockRate;
    return 0;
}

int my_error(int i)
{
    return i % 2;
}

__global__ static void sumOfSquares(int *nums, int *result, clock_t *time)
{
    int sum = 0;
    clock_t start = clock();
    for (int i = 0; i < N; i++)
    {
        sum += nums[i] * nums[i];
    }
    *time = clock() - start;
    printf("time: %ld\n", *time);
    *result = sum;
}

int cal_square_sum(int clock_rate)
{
    int h_data[N];
    for (int i = 0; i < N; i++)
    {
        h_data[i] = i;
    }
    int *h_dev, *result;
    checkCudaErrors(hipMalloc((void **)&h_dev, sizeof(int) * N));
    checkCudaErrors(hipMemcpy(h_dev, h_data, sizeof(int) * N, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **)&result, sizeof(int)));
    clock_t *dev_time;
    hipMalloc((void **)&dev_time, sizeof(clock_t));
    sumOfSquares<<<1, 2, 2>>>(h_dev, result, dev_time);

    clock_t host_time;
    hipMemcpy(&host_time, dev_time, sizeof(clock_t), hipMemcpyDeviceToHost);

    int sum;
    hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);

    printf("sum : %d, time: %d\n", sum, host_time/clock_rate);
}

int main()
{   
    int clock_rate = 0;
    init_cuda(clock_rate);
    printf("clock rate is %d\n", clock_rate);
    cal_square_sum(clock_rate);
    return 0;
}